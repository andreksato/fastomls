#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#define inf 85000

//	const int N = 1520;
//	const int M = 1880;
//	int S[N*M] = {0};
//	float D[N*M] = {0};
    int *dev_S = 0;
    int *dev_P = 0;
    int *dev_P2 = 0;
    float *dev_D = 0;

__global__ void initializationKernel(int * S, int * P, int * P2, int N, int M){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	if(col<M && row<N){
		if (S[row*M+col]==1){
			P[row*M+col]=col;
			P[N*M+row*M+col]=row;
			P2[row*M+col]=col;
			P2[N*M+row*M+col]=row;
		} else{
			P[row*M+col]=inf;
			P[N*M+row*M+col]=inf;
			P2[row*M+col]=inf;
			P2[N*M+row*M+col]=inf;
		}
	}
}

__global__ void distanceKernel(float * D, int * P, int N, int M){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	if(col<M && row<N){
		D[row*M+col]=sqrtf((P[row*M+col]-col)*(P[row*M+col]-col)+(P[N*M+row*M+col]-row)*(P[N*M+row*M+col]-row));
	}
}

__global__ void copy1ColKernel(int * dev_P, int * dev_P2, int N, int M, int col){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	//int row = blockDim.x*threadIdx.y + threadIdx.x;
	if(row<N){
		dev_P2[row*M+col]=dev_P[row*M+col];
		dev_P2[N*M+row*M+col]=dev_P[N*M+row*M+col];
	}
}

__global__ void copyColKernel(int * dev_P, int * dev_P2, int N, int M){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	if(col<M && row<N){
		if(col%2==0){
			dev_P[row*M+col]=dev_P2[row*M+col];
			dev_P[N*M+row*M+col]=dev_P2[N*M+row*M+col];
		} else{
			dev_P2[row*M+col]=dev_P[row*M+col];
			dev_P2[N*M+row*M+col]=dev_P[N*M+row*M+col];
		}
	}
}

__global__ void copy1RowKernel(int * dev_P, int * dev_P2, int N, int M, int row){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	//int col = blockDim.x*threadIdx.y + threadIdx.x;
	if(col<M){
		dev_P2[row*M+col]=dev_P[row*M+col];
		dev_P2[N*M+row*M+col]=dev_P[N*M+row*M+col];
	}
}

__global__ void copyRowKernel(int * dev_P, int * dev_P2, int N, int M){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	if(col<M && row<N){
		if(row%2==0){
			dev_P[row*M+col]=dev_P2[row*M+col];
			dev_P[N*M+row*M+col]=dev_P2[N*M+row*M+col];
		} else{
			dev_P2[row*M+col]=dev_P[row*M+col];
			dev_P2[N*M+row*M+col]=dev_P[N*M+row*M+col];
		}
	}
}

__global__ void propagation1aKernel(int * dev_P, int * dev_P2, int N, int M, int col){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	//int row =blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(row<N){
		L[0] = (dev_P[row*M+col]-col)*(dev_P[row*M+col]-col) + (dev_P[N*M+row*M+col]-row)*(dev_P[N*M+row*M+col]-row);
		Lmin = L[0];
		p = 0;
		L[2] = (dev_P[row*M+col-1]-col)*(dev_P[row*M+col-1]-col) + (dev_P[N*M+row*M+col-1]-row)*(dev_P[N*M+row*M+col-1]-row);
		if (L[2]<Lmin){
			Lmin = L[2];
			p = 2;
		}
		if(row>0){
			L[1] = (dev_P[(row-1)*M+(col-1)]-col)*(dev_P[(row-1)*M+(col-1)]-col) + (dev_P[N*M+(row-1)*M+(col-1)]-row)*(dev_P[N*M+(row-1)*M+(col-1)]-row);
			if (L[1]<Lmin){
				Lmin = L[1];
				p = 1;
			}
		}
		if(row<(N-1)){
			L[3] = (dev_P[(row+1)*M+(col-1)]-col)*(dev_P[(row+1)*M+(col-1)]-col) + (dev_P[N*M+(row+1)*M+(col-1)]-row)*(dev_P[N*M+(row+1)*M+(col-1)]-row);
			if (L[3]<Lmin){
				p = 3;
				Lmin = L[3];
			}
		}
		switch (p){
			case 0:
				dev_P2[row*M+col] = dev_P[row*M+col];
				dev_P2[N*M+row*M+col] = dev_P[N*M+row*M+col];
				break;
			case 1:
				dev_P2[row*M+col] = dev_P[(row-1)*M+(col-1)];
				dev_P2[N*M+row*M+col] = dev_P[N*M+(row-1)*M+(col-1)];
				break;
			case 2:
				dev_P2[row*M+col] = dev_P[row*M+col-1];
				dev_P2[N*M+row*M+col] = dev_P[N*M+row*M+col-1];
				break;
			case 3:
				dev_P2[row*M+col] = dev_P[(row+1)*M+(col-1)];
				dev_P2[N*M+row*M+col] = dev_P[N*M+(row+1)*M+(col-1)];
				break;
		}
	}
}
__global__ void propagation1bKernel(int * dev_P, int * dev_P2, int N, int M, int col){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	//int row =blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(row<N){
		col++;	
		if(col<M){
			L[0] = (dev_P2[row*M+col]-col)*(dev_P2[row*M+col]-col) + (dev_P2[N*M+row*M+col]-row)*(dev_P2[N*M+row*M+col]-row);
			Lmin = L[0];
			p = 0;
			L[2] = (dev_P2[row*M+col-1]-col)*(dev_P2[row*M+col-1]-col) + (dev_P2[N*M+row*M+col-1]-row)*(dev_P2[N*M+row*M+col-1]-row);
			if (L[2]<Lmin){
				Lmin = L[2];
				p = 2;
			}
			if(row>0){
				L[1] = (dev_P2[(row-1)*M+(col-1)]-col)*(dev_P2[(row-1)*M+(col-1)]-col) + (dev_P2[N*M+(row-1)*M+(col-1)]-row)*(dev_P2[N*M+(row-1)*M+(col-1)]-row);
				if (L[1]<Lmin){
					Lmin = L[1];
					p = 1;
				}
			}
			if(row<(N-1)){
				L[3] = (dev_P2[(row+1)*M+(col-1)]-col)*(dev_P2[(row+1)*M+(col-1)]-col) + (dev_P2[N*M+(row+1)*M+(col-1)]-row)*(dev_P2[N*M+(row+1)*M+(col-1)]-row);
				if (L[3]<Lmin){
					Lmin = L[3];
					p = 3;
				}
			}
			switch (p){
				case 0:
					dev_P[row*M+col] = dev_P2[row*M+col];
					dev_P[N*M+row*M+col] = dev_P2[N*M+row*M+col];
					break;
				case 1:
					dev_P[row*M+col] = dev_P2[(row-1)*M+(col-1)];
					dev_P[N*M+row*M+col] = dev_P2[N*M+(row-1)*M+(col-1)];
					break;
				case 2:
					dev_P[row*M+col] = dev_P2[row*M+col-1];
					dev_P[N*M+row*M+col] = dev_P2[N*M+row*M+col-1];
					break;
				case 3:
					dev_P[row*M+col] = dev_P2[(row+1)*M+(col-1)];
					dev_P[N*M+row*M+col] = dev_P2[N*M+(row+1)*M+(col-1)];
					break;
			}
		}
	}
}

__global__ void propagation2aKernel(int * dev_P, int * dev_P2, int N, int M, int col){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	//int row = blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(row<N){
		L[0] = (dev_P2[row*M+col]-col)*(dev_P2[row*M+col]-col) + (dev_P2[N*M+row*M+col]-row)*(dev_P2[N*M+row*M+col]-row);
		Lmin = L[0];
		p = 0;
		L[2] = (dev_P2[row*M+col+1]-col)*(dev_P2[row*M+col+1]-col) + (dev_P2[N*M+row*M+col+1]-row)*(dev_P2[N*M+row*M+col+1]-row);
		if (L[2]<Lmin){
			Lmin = L[2];
			p = 2;
		}
		if(row>0){
			L[1] = (dev_P2[(row-1)*M+(col+1)]-col)*(dev_P2[(row-1)*M+(col+1)]-col) + (dev_P2[N*M+(row-1)*M+(col+1)]-row)*(dev_P2[N*M+(row-1)*M+(col+1)]-row);
			if (L[1]<Lmin){
				Lmin = L[1];
				p = 1;
			}
		}
		if(row<(N-1)){
			L[3] = (dev_P2[(row+1)*M+(col+1)]-col)*(dev_P2[(row+1)*M+(col+1)]-col) + (dev_P2[N*M+(row+1)*M+(col+1)]-row)*(dev_P2[N*M+(row+1)*M+(col+1)]-row);
			if (L[3]<Lmin){
				Lmin = L[3];
				p = 3;
			}
		}
		switch (p){
			case 0:
				dev_P[row*M+col] = dev_P2[row*M+col];
				dev_P[N*M+row*M+col] = dev_P2[N*M+row*M+col];
				break;
			case 1:
				dev_P[row*M+col] = dev_P2[(row-1)*M+(col+1)];
				dev_P[N*M+row*M+col] = dev_P2[N*M+(row-1)*M+(col+1)];
				break;
			case 2:
				dev_P[row*M+col] = dev_P2[row*M+col+1];
				dev_P[N*M+row*M+col] = dev_P2[N*M+row*M+col+1];
				break;
			case 3:
				dev_P[row*M+col] = dev_P2[(row+1)*M+(col+1)];
				dev_P[N*M+row*M+col] = dev_P2[N*M+(row+1)*M+(col+1)];
				break;
		}
	}
}
__global__ void propagation2bKernel(int * dev_P, int * dev_P2, int N, int M, int col){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	//int row = blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(row<N){
		col--;
		if(col>=0){
			L[0] = (dev_P[row*M+col]-col)*(dev_P[row*M+col]-col) + (dev_P[N*M+row*M+col]-row)*(dev_P[N*M+row*M+col]-row);
			Lmin = L[0];
			p = 0;
			L[2] = (dev_P[row*M+col+1]-col)*(dev_P[row*M+col+1]-col) + (dev_P[N*M+row*M+col+1]-row)*(dev_P[N*M+row*M+col+1]-row);
			if (L[2]<Lmin){
				Lmin = L[2];
				p = 2;
			}
			if(row>0){
				L[1] = (dev_P[(row-1)*M+(col+1)]-col)*(dev_P[(row-1)*M+(col+1)]-col) + (dev_P[N*M+(row-1)*M+(col+1)]-row)*(dev_P[N*M+(row-1)*M+(col+1)]-row);
				if (L[1]<Lmin){
					Lmin = L[1];
					p = 1;
				}
			}
			if(row<(N-1)){
				L[3] = (dev_P[(row+1)*M+(col+1)]-col)*(dev_P[(row+1)*M+(col+1)]-col) + (dev_P[N*M+(row+1)*M+(col+1)]-row)*(dev_P[N*M+(row+1)*M+(col+1)]-row);
				if (L[3]<Lmin){
					Lmin = L[3];
					p = 3;
				}
			}
			switch (p){
				case 0:
					dev_P2[row*M+col] = dev_P[row*M+col];
					dev_P2[N*M+row*M+col] = dev_P[N*M+row*M+col];
					break;
				case 1:
					dev_P2[row*M+col] = dev_P[(row-1)*M+(col+1)];
					dev_P2[N*M+row*M+col] = dev_P[N*M+(row-1)*M+(col+1)];
					break;
				case 2:
					dev_P2[row*M+col] = dev_P[row*M+col+1];
					dev_P2[N*M+row*M+col] = dev_P[N*M+row*M+col+1];
					break;
				case 3:
					dev_P2[row*M+col] = dev_P[(row+1)*M+(col+1)];
					dev_P2[N*M+row*M+col] = dev_P[N*M+(row+1)*M+(col+1)];
					break;
			}
		}
	}
}

__global__ void propagation3aKernel(int * dev_P, int * dev_P2, int N, int M, int row){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	//int col = blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(col<M){
		L[0] = (dev_P[row*M+col]-col)*(dev_P[row*M+col]-col) + (dev_P[N*M+row*M+col]-row)*(dev_P[N*M+row*M+col]-row);
		Lmin = L[0];
		p = 0;
		L[2] = (dev_P[(row-1)*M+col]-col)*(dev_P[(row-1)*M+col]-col) + (dev_P[N*M+(row-1)*M+col]-row)*(dev_P[N*M+(row-1)*M+col]-row);
		if (L[2]<Lmin){
			Lmin = L[2];
			p = 2;
		}
		if(col>0){
			L[1] = (dev_P[(row-1)*M+(col-1)]-col)*(dev_P[(row-1)*M+(col-1)]-col) + (dev_P[N*M+(row-1)*M+(col-1)]-row)*(dev_P[N*M+(row-1)*M+(col-1)]-row);
			if (L[1]<Lmin){
				Lmin = L[1];
				p = 1;
			}
		}
		if(col<(M-1)){
			L[3] = (dev_P[(row-1)*M+(col+1)]-col)*(dev_P[(row-1)*M+(col+1)]-col) + (dev_P[N*M+(row-1)*M+(col+1)]-row)*(dev_P[N*M+(row-1)*M+(col+1)]-row);
			if (L[3]<Lmin){
				Lmin = L[3];
				p = 3;
			}
		}	
		switch (p){
			case 0:
				dev_P2[row*M+col] = dev_P[row*M+col];
				dev_P2[N*M+row*M+col] = dev_P[N*M+row*M+col];
				break;
			case 1:
				dev_P2[row*M+col] = dev_P[(row-1)*M+(col-1)];
				dev_P2[N*M+row*M+col] = dev_P[N*M+(row-1)*M+(col-1)];
				break;
			case 2:
				dev_P2[row*M+col] = dev_P[(row-1)*M+col];
				dev_P2[N*M+row*M+col] = dev_P[N*M+(row-1)*M+col];
				break;
			case 3:
				dev_P2[row*M+col] = dev_P[(row-1)*M+(col+1)];
				dev_P2[N*M+row*M+col] = dev_P[N*M+(row-1)*M+(col+1)];
				break;
		}
	}
}
__global__ void propagation3bKernel(int * dev_P, int * dev_P2, int N, int M, int row){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	//int col = blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(col<M){
		row++;
		if(row<N){
			L[0] = (dev_P2[row*M+col]-col)*(dev_P2[row*M+col]-col) + (dev_P2[N*M+row*M+col]-row)*(dev_P2[N*M+row*M+col]-row);
			Lmin = L[0];
			p = 0;
			L[2] = (dev_P2[(row-1)*M+col]-col)*(dev_P2[(row-1)*M+col]-col) + (dev_P2[N*M+(row-1)*M+col]-row)*(dev_P2[N*M+(row-1)*M+col]-row);
			if (L[2]<Lmin){
				Lmin = L[2];
				p = 2;
			}
			if(col>0){
				L[1] = (dev_P2[(row-1)*M+(col-1)]-col)*(dev_P2[(row-1)*M+(col-1)]-col) + (dev_P2[N*M+(row-1)*M+(col-1)]-row)*(dev_P2[N*M+(row-1)*M+(col-1)]-row);
				if (L[1]<Lmin){
					Lmin = L[1];
					p = 1;
				}
			}
			if(col<(M-1)){
				L[3] = (dev_P2[(row-1)*M+(col+1)]-col)*(dev_P2[(row-1)*M+(col+1)]-col) + (dev_P2[N*M+(row-1)*M+(col+1)]-row)*(dev_P2[N*M+(row-1)*M+(col+1)]-row);
				if (L[3]<Lmin){
					Lmin = L[3];
					p = 3;
				}
			}
			switch (p){
				case 0:
					dev_P[row*M+col] = dev_P2[row*M+col];
					dev_P[N*M+row*M+col] = dev_P2[N*M+row*M+col];
					break;
				case 1:
					dev_P[row*M+col] = dev_P2[(row-1)*M+(col-1)];
					dev_P[N*M+row*M+col] = dev_P2[N*M+(row-1)*M+(col-1)];
					break;
				case 2:
					dev_P[row*M+col] = dev_P2[(row-1)*M+col];
					dev_P[N*M+row*M+col] = dev_P2[N*M+(row-1)*M+col];
					break;
				case 3:
					dev_P[row*M+col] = dev_P2[(row-1)*M+(col+1)];
					dev_P[N*M+row*M+col] = dev_P2[N*M+(row-1)*M+(col+1)];
					break;
			}
		}
	}
}
	
__global__ void propagation4aKernel(int * dev_P, int * dev_P2, int N, int M, int row){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	//int col = blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(col<M){
		L[0] = (dev_P2[row*M+col]-col)*(dev_P2[row*M+col]-col) + (dev_P2[N*M+row*M+col]-row)*(dev_P2[N*M+row*M+col]-row);
		Lmin = L[0];
		p =0;
		L[2] = (dev_P2[(row+1)*M+col]-col)*(dev_P2[(row+1)*M+col]-col) + (dev_P2[N*M+(row+1)*M+col]-row)*(dev_P2[N*M+(row+1)*M+col]-row);
		if (L[2]<Lmin){
			Lmin = L[2];
			p = 2;
		}
		if(col>0){
			L[1] = (dev_P2[(row+1)*M+col-1]-col)*(dev_P2[(row+1)*M+col-1]-col) + (dev_P2[N*M+(row+1)*M+col-1]-row)*(dev_P2[N*M+(row+1)*M+col-1]-row);
			if (L[1]<Lmin){
				Lmin = L[1];
				p = 1;
			}
		}
		if(col<(M-1)){
			L[3] = (dev_P2[(row+1)*M+(col+1)]-col)*(dev_P2[(row+1)*M+(col+1)]-col) + (dev_P2[N*M+(row+1)*M+(col+1)]-row)*(dev_P2[N*M+(row+1)*M+(col+1)]-row);
			if (L[3]<Lmin){
				Lmin = L[3];
				p = 3;
			}
		}
		switch (p){
			case 0:
				dev_P[row*M+col] = dev_P2[row*M+col];
				dev_P[N*M+row*M+col] = dev_P2[N*M+row*M+col];
				break;
			case 1:
				dev_P[row*M+col] = dev_P2[(row+1)*M+col-1];
				dev_P[N*M+row*M+col] = dev_P2[N*M+(row+1)*M+col-1];
				break;
			case 2:
				dev_P[row*M+col] = dev_P2[(row+1)*M+col];
				dev_P[N*M+row*M+col] = dev_P2[N*M+(row+1)*M+col];
				break;
			case 3:
				dev_P[row*M+col] = dev_P2[(row+1)*M+(col+1)];
				dev_P[N*M+row*M+col] = dev_P2[N*M+(row+1)*M+(col+1)];
				break;
		}
	}
}
__global__ void propagation4bKernel(int * dev_P, int * dev_P2, int N, int M, int row){
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	//int col = blockDim.x*threadIdx.y + threadIdx.x;
	int L[4];
	int Lmin;
	int p;
	if(col<M){
		row--;
		if(row>=0){
			L[0] = (dev_P[row*M+col]-col)*(dev_P[row*M+col]-col) + (dev_P[N*M+row*M+col]-row)*(dev_P[N*M+row*M+col]-row);
			Lmin = L[0];
			p = 0;
			L[2] = (dev_P[(row+1)*M+col]-col)*(dev_P[(row+1)*M+col]-col) + (dev_P[N*M+(row+1)*M+col]-row)*(dev_P[N*M+(row+1)*M+col]-row);
			if (L[2]<Lmin){
				Lmin = L[2];
				p = 2;
			}
			if(col>0){
				L[1] = (dev_P[(row+1)*M+col-1]-col)*(dev_P[(row+1)*M+col-1]-col) + (dev_P[N*M+(row+1)*M+col-1]-row)*(dev_P[N*M+(row+1)*M+col-1]-row);
				if (L[1]<Lmin){
					Lmin = L[1];
					p = 1;
				}
			}
			if(col<(M-1)){
				L[3] = (dev_P[(row+1)*M+(col+1)]-col)*(dev_P[(row+1)*M+(col+1)]-col) + (dev_P[N*M+(row+1)*M+(col+1)]-row)*(dev_P[N*M+(row+1)*M+(col+1)]-row);
				if (L[3]<Lmin){
					Lmin = L[3];
					p = 3;
				}
			}
			switch (p){
				case 0:
					dev_P2[row*M+col] = dev_P[row*M+col];
					dev_P2[N*M+row*M+col] = dev_P[N*M+row*M+col];
					break;
				case 1:
					dev_P2[row*M+col] = dev_P[(row+1)*M+col-1];
					dev_P2[N*M+row*M+col] = dev_P[N*M+(row+1)*M+col-1];
					break;
				case 2:
					dev_P2[row*M+col] = dev_P[(row+1)*M+col];
					dev_P2[N*M+row*M+col] = dev_P[N*M+(row+1)*M+col];
					break;
				case 3:
					dev_P2[row*M+col] = dev_P[(row+1)*M+(col+1)];
					dev_P2[N*M+row*M+col] = dev_P[N*M+(row+1)*M+(col+1)];
					break;
			}
		}
	}
	__syncthreads();
}

//hipError_t transform( int * S, float * D, int N, int M);

hipError_t transform(int * S, float * D, int N, int M){
	hipError_t cudaStatus;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for vectors
    cudaStatus = hipMalloc((void**)&dev_S, (N*M) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_P, 2*N*M*sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_P2, 2*N*M*sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_D, N*M*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_S, S, N*M*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy HtD failed!");
        goto Error;
    }

	//dim3 dimBlock1(256, ceil(N/256.0), 1);
	//dim3 dimBlock2(256, ceil(M/256.0), 1);
	//dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid1(ceil(N/1024.0));
	dim3 dimGrid2(ceil(M/1024.0));
	dim3 dimBlock3(ceil(M/256.0), ceil(N/256.0), 1);
	dim3 dimGrid3(256, 256, 1);

	hipEventRecord(start, 0);

	initializationKernel<<<dimGrid3, dimBlock3>>>(dev_S, dev_P, dev_P2, N, M);
	
	int i;
	for(i=1; i<M;i+=2){
		propagation1aKernel<<<dimGrid1, dimBlock>>>(dev_P, dev_P2, N, M, i);
		propagation1bKernel<<<dimGrid1, dimBlock>>>(dev_P, dev_P2, N, M, i);
	}
	if(M%2==1){
		copy1ColKernel<<<dimGrid1, dimBlock>>>(dev_P, dev_P2, N, M, M-1);
		for(i=M-2; i>=0;i-=2){
			propagation2aKernel<<<dimGrid1, dimBlock>>>(dev_P, dev_P2, N, M, i);
			propagation2bKernel<<<dimGrid1, dimBlock>>>(dev_P, dev_P2, N, M, i);
		}
	} else{
		copy1ColKernel<<<dimGrid1, dimBlock>>>(dev_P2, dev_P, N, M, M-1);
		for(i=M-2; i>=0;i-=2){
			propagation2aKernel<<<dimGrid1, dimBlock>>>(dev_P2, dev_P, N, M, i);
			propagation2bKernel<<<dimGrid1, dimBlock>>>(dev_P2, dev_P, N, M, i);
		}
	}
	copyColKernel<<<dimGrid3, dimBlock3>>>(dev_P, dev_P2, N, M);
	
	int j;
	for(j=1; j<N;j+=2){
		propagation3aKernel<<<dimGrid2, dimBlock>>>(dev_P, dev_P2, N, M, j);
		propagation3bKernel<<<dimGrid2, dimBlock>>>(dev_P, dev_P2, N, M, j);
	}
	if(N%2==1){
		copy1RowKernel<<<dimGrid2, dimBlock>>>(dev_P, dev_P2, N, M, N-1);
		for(j=N-2; j>=0;j-=2){
			propagation4aKernel<<<dimGrid2, dimBlock>>>(dev_P, dev_P2, N, M, j);
			propagation4bKernel<<<dimGrid2, dimBlock>>>(dev_P, dev_P2, N, M, j);
		}
	} else{
		copy1RowKernel<<<dimGrid2, dimBlock>>>(dev_P2, dev_P, N, M, N-1);
		for(j=N-2; j>=0; j-=2){
			propagation4aKernel<<<dimGrid2, dimBlock>>>(dev_P2, dev_P, N, M, j);
			propagation4bKernel<<<dimGrid2, dimBlock>>>(dev_P2, dev_P, N, M, j);
		}
	}
	copyRowKernel<<<dimGrid3, dimBlock3>>>(dev_P, dev_P2, N, M);
	
	distanceKernel<<<dimGrid3, dimBlock3>>>(dev_D, dev_P, N, M);
	
	cudaStatus = hipDeviceSynchronize();
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

	cudaStatus = hipMemcpy(D, dev_D, N*M*sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy DtH failed!");
        goto Error;
    }

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
//    printf("Execution Time: %f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);

Error:
    hipFree(dev_S);
    hipFree(dev_P);
	hipFree(dev_P2);
	hipFree(dev_D);
	return cudaStatus;
}
